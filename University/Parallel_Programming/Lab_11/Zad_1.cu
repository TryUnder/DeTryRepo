
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void emptyKernel() {}

__global__ void add(int* a, int* b, int* c) {
	c[blockIdx.x]  = a[blockIdx.x] + b[blockIdx.x];
}

int main() {
	unsigned int N;
	std::cout << "Podaj rozmiar" << std::endl;
	std::cin >> N;
	unsigned int size = N;
	int* a;
	int* src1 = new int[size];
	int* b;
	int* src2 = new int[size];
	int* c;
	int* src3 = new int[size];
	int* result = new int[size];
	
	int* cpuSum = new int[size];
	
	hipMalloc((void **)&a, size * sizeof(int));
	hipMalloc((void **)&b, size * sizeof(int));
	hipMalloc((void **)&c, size * sizeof(int));
	
	for (int i = 0; i < size; ++i) {
		src1[i] = i;
		src2[i] = 10;
		src3[i] = 0;
	}
	
	for (int i = 0; i < size; ++i) {
		cpuSum[i] = src1[i] + src2[i];
	}
	
	std::cout << "Suma na CPU: \n" << std::endl;
	
	for (int i = 0; i < size; ++i) {
		std::cout << cpuSum[i] << std::endl;
	}
	
	hipMemcpy(a, src1, sizeof(int) * size, hipMemcpyHostToDevice); 
	hipMemcpy(b, src2, sizeof(int) * size, hipMemcpyHostToDevice);
	hipMemcpy(c, src3, sizeof(int) * size, hipMemcpyHostToDevice);
	
	
	add<<<N,1>>>(a, b, c);
	
	hipMemcpy(result, c, sizeof(int) * size, hipMemcpyDeviceToHost);
	std::cout << "Suma na GPU: \n" << std::endl;
	for (int i = 0; i < size; ++i) {
		std::cout << result[i] << std::endl;
	}
	
	return 0;
}
