#include "hip/hip_runtime.h"
#include <iostream>
#include <omp.h>

double function(double x){
    return (sqrt(1 - pow(x,2)));
}

int main() {
	unsigned int size = 100;
	double tBegin, tEnd;
	double pole = 0.0;
	unsigned int n = 1000000;
	
	double xp = -1.0 + 2.0 / size * rank;
    double xk = -1.0 + 2.0 / size * (rank + 1);
    double dx = (xk - xp) / n;
    
    tBegin = MPI_Wtime();
    for (int i = 0; i < size; ++i) {
			
	}  
}
